#include "hip/hip_runtime.h"
#include "core.h"

#include <stdio.h>
#include <assert.h>
#include <algorithm>
#include <hip/hip_runtime_api.h>
#include <device_atomic_functions.h>
#include <>

#define WLL 1024
#define WL 512
#define W 64
#define H 16

__global__ void kernel_fill_gather(const float *xs, const int *ys, const unsigned int *xn, const unsigned int *yn,
                                   float *gather_xs, long *loc, const unsigned int *memPref, const unsigned int *labelPref,
                                   unsigned int V, unsigned int blank)
{
    unsigned int t = blockIdx.x * W + threadIdx.x;
    unsigned int u = blockIdx.y * H + threadIdx.y;
    unsigned int n = blockIdx.z;

    unsigned int actual_t = xn[n];
    unsigned int actual_u = yn[n] + 1;

    if (t >= actual_t || u >= actual_u)
        return;

    unsigned int mem_loc = memPref[n];

    // l = ys(n, u)
    unsigned int _index = mem_loc + t * actual_u + u;
    float *ptr_gather = gather_xs + (_index << 1);
    // gather_xs(n, t, u, 0) = xs(n, t, u, blank)
    *(ptr_gather++) = xs[_index * V + blank];
    unsigned int l;
    if (u == yn[n])
    {
        // last row
        l = blank;
    }
    else
    {
        l = ys[labelPref[n] + u];
    }
    loc[_index] = l;
    // gather_xs(n, t, u, 1) = xs(n, t, u, l)
    *ptr_gather = xs[_index * V + l];
}

rnntStatus_t run_gather(hipStream_t stream, const float *xs, const int *ys, const unsigned int *xn, const unsigned int *yn,
                        float *gather_xs, long *loc,
                        const unsigned int *memPref, const unsigned int *labelPref,
                        unsigned int N, unsigned int T, unsigned int U, unsigned int V, unsigned int blank)
{

    dim3 threads1(W, H);
    dim3 blocks1((T + W - 1) / W, (U + H - 1) / H, N);

    kernel_fill_gather<<<blocks1, threads1, 0, stream>>>(xs, ys, xn, yn, gather_xs, loc, memPref, labelPref, V, blank);
    if (hipGetLastError() != hipSuccess)
        return RNNT_STATUS_WARP_FAILED;

    return RNNT_STATUS_SUCCESS;
}

__global__ void kernel_fill_scatter_grad(const float *grad_cost, const float *gather_grad, const long *loc,
                                         const unsigned int *cumSum, float *scatter_grad,
                                         unsigned int STU, unsigned int V, unsigned int blank)
{
    // return;
    // unsigned int i = blockIdx.x * WL + threadIdx.x;
    unsigned int i = blockIdx.y * (WL * gridDim.x) + blockIdx.x * WL + threadIdx.x;
    if (i >= STU)
        return;

    // unsigned int n = blockIdx.y;
    unsigned int n = blockIdx.z;
    // ensure i in [cumSum[n-1], cumSum[n]]
    if (i >= cumSum[n] || (n > 0 && i < cumSum[n - 1]))
        return;

    if (threadIdx.y == 0)
    {
        // fill blank label grad
        scatter_grad[i * V + blank] = gather_grad[i << 1] * grad_cost[n];
    }
    else //if (threadIdx.y == 1)
    {
        // fill real label grad
        if (loc[i] > 0)
            scatter_grad[i * V + loc[i]] = gather_grad[(i << 1) + 1] * grad_cost[n];
    }
}

rnntStatus_t run_scatter_grad(hipStream_t stream, const float *grad_cost, const float *gather_grad,
                              const long *loc, const unsigned int *cumSum,
                              float *scatter_grad, unsigned int STU, unsigned int N, unsigned int V, unsigned int blank)
{
    // grad_cost (N, )
    // gather_grad (STU, 2)
    // scatter_grad (STU, V)

    dim3 threads1(WL, 2);

    // dim3 blocks1((STU + WL - 1) / WL, N);

    // STU/WL = (STU/WL)/W, W = (1 + (STU - 1)/WL)/W, W = (1 + (1 + (STU - 1)/WL) - 1)/W, W
    dim3 blocks1(1 + ((1 + (STU - 1) / WL) - 1) / W, W, N);

    kernel_fill_scatter_grad<<<blocks1, threads1, 0, stream>>>(grad_cost, gather_grad, loc, cumSum, scatter_grad, STU, V, blank);
    if (hipGetLastError() != hipSuccess)
        return RNNT_STATUS_WARP_FAILED;

    return RNNT_STATUS_SUCCESS;
}

__global__ void kernel_fill_grad(const float *grad_cost, float *grad, const unsigned int *cumSum, unsigned int STU, unsigned int V)
{

    unsigned int i = blockIdx.x * W + threadIdx.x;
    if (i >= STU)
        return;

    unsigned int n = blockIdx.z;
    // ensure i in [cumSum[n-1], cumSum[n]]
    if (i >= cumSum[n] || (n > 0 && i < cumSum[n - 1]))
        return;

    unsigned int v = blockIdx.y * H + threadIdx.y;
    if (v >= V)
        return;

    grad[i * V + v] *= grad_cost[n];
}

rnntStatus_t run_backward_compact(hipStream_t stream, const float *grad_cost, float *grad,
                                  const unsigned int *cumSum, unsigned int STU, unsigned N, unsigned int V)
{
    // grad_cost (N, )
    // grad (STU, V)
    dim3 threads(W, H);
    dim3 blocks((STU + W - 1) / W, (V + H - 1) / H, N); // (N-1) redundancy

    kernel_fill_grad<<<blocks, threads, 0, stream>>>(grad_cost, grad, cumSum, STU, V);
    if (hipGetLastError() != hipSuccess)
        return RNNT_STATUS_WARP_FAILED;

    return RNNT_STATUS_SUCCESS;
}